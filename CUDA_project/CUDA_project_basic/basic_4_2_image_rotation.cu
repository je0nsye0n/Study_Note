#include "hip/hip_runtime.h"
#include <cstdio>

#include "image_rotation.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void rotate_image_kernel(float *input, float *output, int W, int H,
                                    float sin_theta, float cos_theta) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  int dest_x = blockDim.x * blockIdx.x + threadIdx.x;
  int dest_y = blockDim.y * blockIdx.y + threadIdx.y;

  if (dest_x >= W || dest_y >= H) return;

  float xOff = dest_x - x0;
  float yOff = dest_y - y0;
  int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
  int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
  if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
    output[dest_y * W + dest_x] = input[src_y * W + src_x];
  } else {
    output[dest_y * W + dest_x] = 0.0f;
  }
}

// Device(GPU) pointers
static float *input_gpu, *output_gpu;

void rotate_image_naive(float *input_images, float *output_images, int W, int H,
                        float sin_theta, float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  for (int i = 0; i < num_src_images; i++) {
    for (int dest_x = 0; dest_x < W; dest_x++) {
      for (int dest_y = 0; dest_y < H; dest_y++) {
        float xOff = dest_x - x0;
        float yOff = dest_y - y0;
        int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
        int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
        if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
          output_images[i * H * W + dest_y * W + dest_x] =
              input_images[i * H * W + src_y * W + src_x];
        } else {
          output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
        }
      }
    }
  }
}

void rotate_image(float *input_images, float *output_images, int W, int H,
                  float sin_theta, float cos_theta, int num_src_images) {
  for (int i = 0; i < num_src_images; i++) {
    CHECK_CUDA(hipMemcpy(input_gpu, input_images + i * H * W,
                          W * H * sizeof(float), hipMemcpyHostToDevice));
    dim3 block_dim(32, 32);
    dim3 grid_dim((W + block_dim.x - 1) / block_dim.x,
                  (H + block_dim.y - 1) / block_dim.y);
    rotate_image_kernel<<<grid_dim, block_dim>>>(input_gpu, output_gpu, W, H,
                                                 sin_theta, cos_theta);
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(output_images + i * H * W, output_gpu,
                          W * H * sizeof(float), hipMemcpyDeviceToHost));
  }

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_init(int image_width, int image_height, int num_src_images) {
  CHECK_CUDA(
      hipMalloc(&input_gpu, image_width * image_height * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&output_gpu, image_width * image_height * sizeof(float)));

      // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_cleanup() {
  CHECK_CUDA(hipFree(input_gpu));
  CHECK_CUDA(hipFree(output_gpu));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
