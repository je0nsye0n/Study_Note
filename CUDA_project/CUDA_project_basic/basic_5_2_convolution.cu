#include "hip/hip_runtime.h"
#include <cstdio>

#include "convolution.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void convolution_kernel(float *I, float *F, float *O, int N, int C,
                                   int H, int W, int K, int R, int S, int pad_h,
                                   int pad_w, int stride_h, int stride_w,
                                   int dilation_h, int dilation_w) {
  const int ON = N;
  const int OC = K;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;

  // parse (n, c, h, w) from thread index
  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  const int on = tidx / (OC * OH * OW);
  const int oc = (tidx / (OH * OW)) % OC;
  const int oh = (tidx / OW) % OH;
  const int ow = tidx % OW;

  if (on >= ON || oc >= OC || oh >= OH || ow >= OW) return;

  float sum = 0;
  for (int c = 0; c < C; ++c) {
    for (int r = 0; r < R; ++r) {
      for (int s = 0; s < S; ++s) {
        const int n = on;
        const int h = oh * stride_h - pad_h + r * dilation_h;
        const int w = ow * stride_w - pad_w + s * dilation_w;
        const int k = oc;
        if (h < 0 || h >= H || w < 0 || w >= W) continue;
        sum +=
            I[((n * C + c) * H + h) * W + w] * F[((k * C + c) * R + r) * S + s];
      }
    }
  }
  O[((on * OC + oc) * OH + oh) * OW + ow] = sum;
}

float *I_gpu, *F_gpu, *O_gpu;

void naive_cpu_convolution(float *_I, float *_F, float *_O, int N, int C, int H,
                           int W, int K, int R, int S, int pad_h, int pad_w,
                           int stride_h, int stride_w, int dilation_h,
                           int dilation_w) {
  float *I = _I, *F = _F, *O = _O;
  // Naive CPU convolution
  const int ON = N;
  const int OC = K;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;
  for (int on = 0; on < ON; ++on) {
    for (int oc = 0; oc < OC; ++oc) {
      for (int oh = 0; oh < OH; ++oh) {
        for (int ow = 0; ow < OW; ++ow) {
          float sum = 0;
          for (int c = 0; c < C; ++c) {
            for (int r = 0; r < R; ++r) {
              for (int s = 0; s < S; ++s) {
                const int n = on;
                const int h = oh * stride_h - pad_h + r * dilation_h;
                const int w = ow * stride_w - pad_w + s * dilation_w;
                const int k = oc;
                if (h < 0 || h >= H || w < 0 || w >= W) continue;
                sum += I[((n * C + c) * H + h) * W + w] *
                       F[((k * C + c) * R + r) * S + s];
              }
            }
          }
          O[((on * OC + oc) * OH + oh) * OW + ow] = sum;
        }
      }
    }
  }
}

void convolution(float *_I, float *_F, float *_O, int N, int C, int H, int W,
                 int K, int R, int S, int pad_h, int pad_w, int stride_h,
                 int stride_w, int dilation_h, int dilation_w) {
  const int ON = N;
  const int OC = K;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;

  CHECK_CUDA(hipMemcpy(I_gpu, _I, N * C * H * W * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F_gpu, _F, K * C * R * S * sizeof(float),
                        hipMemcpyHostToDevice));

  int total_threads = N * K * OH * OW;
  int block_size = 1024;
  dim3 blockDim(block_size);
  dim3 gridDim((total_threads + block_size - 1) / block_size);
  convolution_kernel<<<gridDim, blockDim>>>(I_gpu, F_gpu, O_gpu, N, C, H, W, K,
                                            R, S, pad_h, pad_w, stride_h,
                                            stride_w, dilation_h, dilation_w);
  CHECK_CUDA(hipGetLastError());

  CHECK_CUDA(hipMemcpy(_O, O_gpu, ON * OC * OH * OW * sizeof(float),
                        hipMemcpyDeviceToHost));
  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void convolution_initialize(int N, int C, int H, int W, int K, int R, int S,
                            int pad_h, int pad_w, int stride_h, int stride_w,
                            int dilation_h, int dilation_w) {
  const int ON = N;
  const int OC = K;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;

  CHECK_CUDA(hipMalloc(&I_gpu, N * C * H * W * sizeof(float)));
  CHECK_CUDA(hipMalloc(&F_gpu, K * C * R * S * sizeof(float)));
  CHECK_CUDA(hipMalloc(&O_gpu, ON * OC * OH * OW * sizeof(float)));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void convolution_cleanup(float *_I, float *_F, float *_O, int N, int C, int H,
                         int W, int K, int R, int S, int pad_h, int pad_w,
                         int stride_h, int stride_w, int dilation_h,
                         int dilation_w) {
  CHECK_CUDA(hipFree(I_gpu));
  CHECK_CUDA(hipFree(F_gpu));
  CHECK_CUDA(hipFree(O_gpu));
  CHECK_CUDA(hipDeviceSynchronize());
}