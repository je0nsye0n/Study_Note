#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"
#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define BLOCK_SIZE 32

static __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                                     int K) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int gj = blockIdx.x, gi = blockIdx.y;
  int lj = threadIdx.x, li = threadIdx.y;
  if (gi * BLOCK_SIZE >= M || gj * BLOCK_SIZE >= N) return;

  __shared__ float Alocal[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Blocal[BLOCK_SIZE][BLOCK_SIZE];
  float c = 0.f;

  for (int bk = 0; bk < K; bk += BLOCK_SIZE) {
    int Ai = gi * BLOCK_SIZE + li, Bj = gj * BLOCK_SIZE + lj;
    int Aj = bk + lj, Bi = bk + li;
    Alocal[li][lj] = (Ai < M && Aj < K) ? A[Ai * K + Aj] : 0.f;
    Blocal[li][lj] = (Bi < K && Bj < N) ? B[Bi * N + Bj] : 0.f;
    __syncthreads();

    for (int lk = 0; lk < BLOCK_SIZE; ++lk) {
      c += Alocal[li][lk] * Blocal[lk][lj];
    }
    __syncthreads();
  }

  if (i < M && j < N) C[i * N + j] = c;
}

static float *A_gpu, *B_gpu, *C_gpu;

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(
      hipMemcpy(A_gpu, _A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(B_gpu, _B, K * N * sizeof(float), hipMemcpyHostToDevice));
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
               (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
  matmul_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(
      hipMemcpy(_C, C_gpu, M * N * sizeof(float), hipMemcpyDeviceToHost));
}

void matmul_init(int M, int N, int K) {
  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));
}