#include "hip/hip_runtime.h"
#include <cstdio>

#include "vecadd.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void vecadd_kernel(const int N, const float *a, const float *b,
                              float *c) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx >= N) return;
  c[tidx] = a[tidx] + b[tidx];
}

#define BLOCKS 8
static int Nbegin[BLOCKS], Nend[BLOCKS];
static hipStream_t upload_stream, download_stream, calc_stream;
static hipEvent_t upload_events[BLOCKS], calc_events[BLOCKS];

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void vecadd(float *_A, float *_B, float *_C, int N) {
  // Upload A and B vector to GPU
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipMemcpyAsync(A_gpu + Nbegin[i], _A + Nbegin[i],
                               (Nend[i] - Nbegin[i]) * sizeof(float),
                               hipMemcpyHostToDevice, upload_stream));
    CHECK_CUDA(hipMemcpyAsync(B_gpu + Nbegin[i], _B + Nbegin[i],
                               (Nend[i] - Nbegin[i]) * sizeof(float),
                               hipMemcpyHostToDevice, upload_stream));
    CHECK_CUDA(hipEventRecord(upload_events[i], upload_stream));
  }

  // Launch kernel on a GPU
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipStreamWaitEvent(calc_stream, upload_events[i]));
    dim3 gridDim((Nend[i] - Nbegin[i] + 512 - 1) / 512);
    dim3 blockDim(512);
    vecadd_kernel<<<gridDim, blockDim, 0, calc_stream>>>(
        Nend[i] - Nbegin[i], A_gpu + Nbegin[i], B_gpu + Nbegin[i],
        C_gpu + Nbegin[i]);
    CHECK_CUDA(hipEventRecord(calc_events[i], calc_stream));
  }

  // Download C vector from GPU
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipStreamWaitEvent(download_stream, calc_events[i]));
    CHECK_CUDA(hipMemcpyAsync(_C + Nbegin[i], C_gpu + Nbegin[i],
                               (Nend[i] - Nbegin[i]) * sizeof(float),
                               hipMemcpyDeviceToHost, download_stream));
  }

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void vecadd_init(int N) {
  for (int i = 0; i < BLOCKS; i++) {
    Nbegin[i] = N / BLOCKS * i;
    Nend[i] = N / BLOCKS * (i + 1);
    if (i == BLOCKS - 1) Nend[i] = N;
  }

  // Allocate device memory
  CHECK_CUDA(hipMalloc(&A_gpu, N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, N * sizeof(float)));

  // Create streams
  CHECK_CUDA(hipStreamCreate(&upload_stream));
  CHECK_CUDA(hipStreamCreate(&download_stream));
  CHECK_CUDA(hipStreamCreate(&calc_stream));
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipEventCreate(&upload_events[i]));
    CHECK_CUDA(hipEventCreate(&calc_events[i]));
  }

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void vecadd_cleanup(float *_A, float *_B, float *_C, int N) {
  // Free GPU memoryy
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  CHECK_CUDA(hipStreamDestroy(upload_stream));
  CHECK_CUDA(hipStreamDestroy(download_stream));
  CHECK_CUDA(hipStreamDestroy(calc_stream));
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipEventDestroy(upload_events[i]));
    CHECK_CUDA(hipEventDestroy(calc_events[i]));
  }

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
