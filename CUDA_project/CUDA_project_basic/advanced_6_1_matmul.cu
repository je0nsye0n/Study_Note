#include "hip/hip_runtime.h"
#include <cstdio>
#include <nvToolsExt.h>
#include <hip/hip_runtime_api.h>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                              int K) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (j >= N || i >= M) return;

  // init C
  C[i * N + j] = 0;

  float a0, a1, a2, a3, a4, a5, a6, a7;
  float b0, b1, b2, b3, b4, b5, b6, b7;
  int k;

  // loop unrolling
  for (k = 0; k + 7 < K; k += 8) {
    a0 = A[i * K + (k + 0)];
    a1 = A[i * K + (k + 1)];
    a2 = A[i * K + (k + 2)];
    a3 = A[i * K + (k + 3)];
    a4 = A[i * K + (k + 4)];
    a5 = A[i * K + (k + 5)];
    a6 = A[i * K + (k + 6)];
    a7 = A[i * K + (k + 7)];
    b0 = B[(k + 0) * N + j];
    b1 = B[(k + 1) * N + j];
    b2 = B[(k + 2) * N + j];
    b3 = B[(k + 3) * N + j];
    b4 = B[(k + 4) * N + j];
    b5 = B[(k + 5) * N + j];
    b6 = B[(k + 6) * N + j];
    b7 = B[(k + 7) * N + j];
    C[i * N + j] += a0 * b0 + a1 * b1 + a2 * b2 + a3 * b3 + a4 * b4 + a5 * b5 +
                    a6 * b6 + a7 * b7;
  }

  // Deal with trailing k
  for (; k < K; k++) {
    C[i * N + j] += A[i * K + k] * B[k * N + j];
  }
}

// Color preset 
unsigned int nvtx_colors[16] = {
  0xFF00FF00, 0xFF008000, 0xFF00FFFF, 0xFF008080, 0xFF0000FF, 0xFF000080,
  0xFFFF00FF, 0xFF800080, 0xFFFFFFFF, 0xFFC0C0C0, 0xFF808080, 0xFF000000,
  0xFFFF0000, 0xFF800000, 0xFFFFFF00, 0xFF808000,
};

void nvtx_add_marker(const char *message) {
  nvtxEventAttributes_t eventAttrib = {0};
  eventAttrib.version = NVTX_VERSION;
  eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  eventAttrib.colorType = NVTX_COLOR_ARGB;
  eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
  eventAttrib.message.ascii = message;
  eventAttrib.color = nvtx_colors[rand() % 16];
  nvtxMarkEx(&eventAttrib);
}

void nvtx_range_push(const char *message) {
  nvtxEventAttributes_t eventAttrib = {0};
  eventAttrib.version = NVTX_VERSION;
  eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  eventAttrib.colorType = NVTX_COLOR_ARGB;
  eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
  eventAttrib.message.ascii = message;
  eventAttrib.color = nvtx_colors[rand() % 16];

  nvtxRangePushEx(&eventAttrib);
}

void nvtx_range_pop() {
  nvtxRangePop();
}

nvtxRangeId_t nvtx_range_start(const char *message) {
  nvtxEventAttributes_t eventAttrib = {0};
  eventAttrib.version = NVTX_VERSION;
  eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  eventAttrib.colorType = NVTX_COLOR_ARGB;
  eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
  eventAttrib.message.ascii = message;
  eventAttrib.color = nvtx_colors[rand() % 16];

  return nvtxRangeStartEx(&eventAttrib);
}

void nvtx_range_stop(nvtxRangeId_t nvtx_id) {
  nvtxRangeEnd(nvtx_id);
}

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  nvtxRangeId_t matmul_id = nvtx_range_start("matmul");
  nvtxRangeId_t id = nvtx_range_start("Upload data");
  CHECK_CUDA(
      hipMemcpy(A_gpu, _A, sizeof(float) * M * K, hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(B_gpu, _B, sizeof(float) * K * N, hipMemcpyHostToDevice));
  nvtx_range_stop(id);

  id = nvtx_range_start("Run kernel");
  dim3 griddim((N + 16 - 1) / 16, (M + 16 - 1) / 16);
  dim3 blockdim(16, 16);
  matmul_kernel<<<griddim, blockdim>>>(A_gpu, B_gpu, C_gpu, M, N, K);
  CHECK_CUDA(hipDeviceSynchronize());
  nvtx_range_stop(id);

  id = nvtx_range_start("Download data");
  CHECK_CUDA(
      hipMemcpy(_C, C_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost));
  nvtx_range_stop(id);

  // Dummy ranges
  nvtx_range_push("Overlapping Range0");
  nvtx_range_push("Overlapping Range1");
  nvtx_range_push("Overlapping Range2");
  nvtx_range_pop();
  nvtx_range_pop();
  nvtx_range_pop();

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());

  // Dummy markers
  nvtxMarkA("End of matmul");
  nvtx_add_marker("End of matmul again");
  nvtx_range_stop(matmul_id);
}

void matmul_init(int M, int N, int K) {
  CHECK_CUDA(hipMalloc((void **) &A_gpu, sizeof(float) * M * K));
  CHECK_CUDA(hipMalloc((void **) &B_gpu, sizeof(float) * K * N));
  CHECK_CUDA(hipMalloc((void **) &C_gpu, sizeof(float) * M * N));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
