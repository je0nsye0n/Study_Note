#include "hip/hip_runtime.h"
#include <cstdio>

#include "image_rotation.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *input_images_gpu, *output_images_gpu;

__global__ void rotate_image_kernel(float *input_images, float *output_images, int W, int H,
                        float sin_theta, float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  int i = blockIdx.z;
  int dest_x = threadIdx.x + blockIdx.x * blockDim.x;
  int dest_y = threadIdx.y + blockIdx.y * blockDim.y;
  if (dest_x > W || dest_y > H) return;

  float xOff = dest_x - x0;
  float yOff = dest_y - y0;
  int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
  int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
  
  if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
    output_images[i * H * W + dest_y * W + dest_x] = input_images[i * H * W + src_y * W + src_x];
  } else {
    output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
  }
}

void rotate_image(float *input_images, float *output_images, int W, int H,
                  float sin_theta, float cos_theta, int num_src_images) {
  // (TODO) Upload input images to GPU
  CHECK_CUDA(hipMemcpy(input_images_gpu, input_images, sizeof(float) * W * H * num_src_images, hipMemcpyHostToDevice));

  // (TODO) Launch kernel on GPU
  dim3 griddim((W+31)/32, (H+31)/32, num_src_images);
  dim3 blockdim(32, 32, 1);
  rotate_image_kernel<<<griddim, blockdim>>>(input_images_gpu, output_images_gpu, W, H, sin_theta, cos_theta, num_src_images);
 
  // (TODO) Download output images from GPU
  CHECK_CUDA(hipMemcpy(output_images, output_images_gpu, sizeof(float) * W * H * num_src_images, hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_init(int image_width, int image_height, int num_src_images) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc((void**)&input_images_gpu, sizeof(float) * num_src_images * image_width * image_height)); 
  CHECK_CUDA(hipMalloc((void**)&output_images_gpu, sizeof(float) * num_src_images * image_width * image_height)); 

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_cleanup() {
  // (TODO) Free device memory
  CHECK_CUDA(hipFree(input_images_gpu));
  CHECK_CUDA(hipFree(output_images_gpu));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
