#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void matmul_kernel(float *A_T, float4 *B, float4 *C, int M, int N, int K) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  if (i >= M || j * 4 >= N) return;
  float4 sum = make_float4(0, 0, 0, 0);
  for (int k = 0; k < K; ++k) {
    float a = A_T[k * M + i];
    float4 b = B[k * (N / 4) + j];
    sum = make_float4(sum.x + a * b.x, sum.y + a * b.y,
                      sum.z + a * b.z, sum.w + a * b.w);
  }
  C[i * (N / 4) + j] = sum;
}

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(
      hipMemcpy(A_gpu, _A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(B_gpu, _B, K * N * sizeof(float), hipMemcpyHostToDevice));

  dim3 blockDim(32, 32);
  dim3 gridDim((M + 32 - 1) / 32, (N / 4 + 32 - 1) / 32);
  matmul_kernel<<<gridDim, blockDim>>>(A_gpu, (float4*)B_gpu, (float4*)C_gpu, M, N, K);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(
      hipMemcpy(_C, C_gpu, M * N * sizeof(float), hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
