#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                              int K) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;

  if (j >= N || i >= M) return;

  // init C
  C[i * N + j] = 0;

  float a0, a1, a2, a3, a4, a5, a6, a7;
  float b0, b1, b2, b3, b4, b5, b6, b7;
  int k;

  // loop unrolling
  for (k = 0; k + 7 < K; k += 8) {
    a0 = A[i * K + (k + 0)];
    a1 = A[i * K + (k + 1)];
    a2 = A[i * K + (k + 2)];
    a3 = A[i * K + (k + 3)];
    a4 = A[i * K + (k + 4)];
    a5 = A[i * K + (k + 5)];
    a6 = A[i * K + (k + 6)];
    a7 = A[i * K + (k + 7)];
    b0 = B[(k + 0) * N + j];
    b1 = B[(k + 1) * N + j];
    b2 = B[(k + 2) * N + j];
    b3 = B[(k + 3) * N + j];
    b4 = B[(k + 4) * N + j];
    b5 = B[(k + 5) * N + j];
    b6 = B[(k + 6) * N + j];
    b7 = B[(k + 7) * N + j];
    C[i * N + j] += a0 * b0 + a1 * b1 + a2 * b2 + a3 * b3 + a4 * b4 + a5 * b5 +
                    a6 * b6 + a7 * b7;
  }

  // Deal with trailing k
  for (; k < K; k++) {
    C[i * N + j] += A[i * K + k] * B[k * N + j];
  }
}

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(
      hipMemcpy(A_gpu, _A, sizeof(float) * M * K, hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(B_gpu, _B, sizeof(float) * K * N, hipMemcpyHostToDevice));

  dim3 griddim((N + 16 - 1) / 16, (M + 16 - 1) / 16);
  dim3 blockdim(16, 16);
  matmul_kernel<<<griddim, blockdim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  CHECK_CUDA(
      hipMemcpy(_C, C_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  CHECK_CUDA(hipMalloc((void **) &A_gpu, sizeof(float) * M * K));
  CHECK_CUDA(hipMalloc((void **) &B_gpu, sizeof(float) * K * N));
  CHECK_CUDA(hipMalloc((void **) &C_gpu, sizeof(float) * M * N));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
