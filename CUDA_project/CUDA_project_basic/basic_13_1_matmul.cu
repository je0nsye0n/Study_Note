#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

static __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                                     int K) {
  int j = blockDim.x * blockIdx.x + threadIdx.x;
  int i = blockDim.y * blockIdx.y + threadIdx.y;
  if (i >= M || j >= N) return;
  float sum = 0.0;
  for (int k = 0; k < K; ++k) sum += A[i * K + k] * B[k * N + j];
  C[i * N + j] = sum;
}

#define NGPU 4
#define EVENTS_PER_GPU 1  // Increase as needed

static int Mbegin[NGPU], Mend[NGPU];
static int ngpu;
static hipStream_t streams[NGPU];
static hipEvent_t events[NGPU][EVENTS_PER_GPU];
static float *A_gpu[NGPU], *B_gpu[NGPU], *C_gpu[NGPU];

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(A_gpu[i], &_A[Mbegin[i] * K],
                               (Mend[i] - Mbegin[i]) * K * sizeof(float),
                               hipMemcpyHostToDevice, streams[i]));
    CHECK_CUDA(hipMemcpyAsync(B_gpu[i], _B, K * N * sizeof(float),
                               hipMemcpyHostToDevice, streams[i]));
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    dim3 blockDim(16, 16);
    dim3 gridDim((N + 16 - 1) / 16, (Mend[i] - Mbegin[i] + 16 - 1) / 16);
    matmul_kernel<<<gridDim, blockDim, 0, streams[i]>>>(
        A_gpu[i], B_gpu[i], C_gpu[i], Mend[i] - Mbegin[i], N, K);
    CHECK_CUDA(hipGetLastError());
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(&_C[Mbegin[i] * N], C_gpu[i],
                               (Mend[i] - Mbegin[i]) * N * sizeof(float),
                               hipMemcpyDeviceToHost, streams[i]));
  }

  for (int i = 0; i < ngpu; i++) {
    hipSetDevice(i);
    hipStreamSynchronize(streams[i]);
  }
}

void matmul_init(int M, int N, int K) {
  ngpu = 4;

  for (int i = 0; i < ngpu; i++) {
    Mbegin[i] = M / ngpu * i;
    Mend[i] = M / ngpu * (i + 1);
    if (i == ngpu - 1) Mend[i] = M;
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
    for (int j = 0; j < EVENTS_PER_GPU; j++) {
      CHECK_CUDA(hipEventCreate(&events[i][j]));
    }
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(
        hipMalloc(&A_gpu[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(
        hipMalloc(&C_gpu[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  }
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipFree(A_gpu[i]));
    CHECK_CUDA(hipFree(B_gpu[i]));
    CHECK_CUDA(hipFree(C_gpu[i]));
    CHECK_CUDA(hipStreamDestroy(streams[i]));
    for (int j = 0; j < EVENTS_PER_GPU; j++) {
      CHECK_CUDA(hipEventDestroy(events[i][j]));
    }
  }
}